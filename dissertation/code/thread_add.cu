
#include <hip/hip_runtime.h>
#include<stdio.h>

//Vector Size
#define N 32

//Device Function
__global__ void add(int* a, int* b, int* c)
{
  //perfrom single addition
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
  //store result in c
}

//Generate N random integers, store in a
void random_ints(int* a)
{

  int i;
  for(i=0; i < N; i++)
  {
    a[i] = rand() % 10;
    printf("%02d ", a[i]);
  }
  printf("\n");
}


int main(void)
{
  //Host Arrays
  int *a, *b, *c;
  //Device Arrays
  int *d_a, *d_b, *d_c;

  //Total mem size
  int size = N * sizeof(int);

  //Allocate device mem
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  a = (int *)malloc(size); random_ints(a);
  b = (int *)malloc(size); random_ints(b);
  //Allocate and populate a,b

  c = (int *)malloc(size);
  //Allocate c

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  //Copy a and b to device memory, store in d_a and d_b

  //Execute in 1 block, N threads
  add<<<1,N>>>(d_a, d_b, d_c);

  //Copy result back from device
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i=0; i < N; i++)
  {
    printf("%02d ", c[i]);
  }
  printf("\n");

  //--Free Memory--//
  free(a);
  free(b);
  free(c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  //---------------//

  return 0;
}
